#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <cassert>
#include <cstdio>
#include <cstdlib>

#include "util.h"

/*
WRITE CUDA KERNEL FOR COUNT HERE
*/
const int CHUNK_SIZE = 32;
const int CHUNK_ROWS = 8;

int serial_implementation(int * data, int rows, int cols) {
    int count = 0;
    for (int i = 0; i < rows * cols; i++) {
        if (data[i] == 1) count++;
    }
    return count;
}

__global__ void matrix_count(int* data, int* count, int* rows, int* cols){
    int x = blockIdx.x * CHUNK_SIZE + threadIdx.x;
    int y = blockIdx.y * CHUNK_SIZE + threadIdx.y;

    for (int i=0; i<CHUNK_SIZE; i+= CHUNK_ROWS){
        if (x < *cols && y+i < *rows) {
            if (data[(y + i) * *cols + x] == 1)
                atomicAdd(count, 1);
        }
    }
}


int main(int argc, char ** argv) {
    
    int rows = 0, cols = 0;

    assert(argc == 2);
    int * data = read_file(argv[1], &rows, &cols);

    hipStream_t stream;
    hipEvent_t begin, end;
    hipStreamCreate(&stream);
    hipEventCreate(&begin);
    hipEventCreate(&end);

    int *count_h = 0; // THIS VARIABLE SHOULD HOLD THE TOTAL COUNT BY THE END

    /*
    PERFORM NECESSARY VARIABLE DECLARATIONS HERE
    PERFORM NECESSARY DATA TRANSFER HERE
    */
    int *rows_p, *cols_p;
    int *data_p;

    hipMallocManaged(&data_p, rows * cols * sizeof(int));
    hipMallocManaged(&count_h, sizeof(int));
    hipMallocManaged(&rows_p, sizeof(int));
    hipMallocManaged(&cols_p, sizeof(int));

    *rows_p = rows;
    *cols_p = cols;
    *count_h = 0;
    for (int i=0; i<rows*cols; i++){
        data_p[i] = data[i];
    }

    hipEventRecord(begin, stream);

    /*
    LAUNCH KERNEL HERE
    */
    size_t thread_x = CHUNK_SIZE;
    size_t thread_y = CHUNK_SIZE;
    // ceiling of cols/threads_x
    size_t grid_x = (cols + thread_x - 1) / thread_x;
    // ceiling of rows/threads_y
    size_t grid_y = (rows + thread_y - 1) / thread_y;

    dim3 grid_dim(grid_x, grid_y, 1);
    dim3 block_dim(CHUNK_SIZE, CHUNK_ROWS, 1);

    matrix_count <<<grid_dim, block_dim>>> (data_p, count_h, rows_p, cols_p);
    hipDeviceSynchronize();

    hipEventRecord(end, stream);
    /*
    PERFORM NECESSARY DATA TRANSFER HERE
    */

    hipStreamSynchronize(stream);


    float ms;
    hipEventElapsedTime(&ms, begin, end);
    printf("Elapsed time: %f ms\n", ms);

    /*
    DEALLOCATE RESOURCES HERE
    */
    int count_serial = serial_implementation(data, rows, cols);
    if (count_serial != *count_h) {
        printf("ERROR: %d != %d\n", count_serial, *count_h);
    }

    hipEventDestroy(begin);
    hipEventDestroy(end);
    hipStreamDestroy(stream);

    free(data);
    hipFree(data);
    hipFree(rows_p);
    hipFree(cols_p);

    return 0;
}